#define _USE_MATH_DEFINES

#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

const int threads_total = 4;
__device__ hiprandState_t* curand_states[threads_total];

__constant__ int steps_per_kernel_call = 200;
__constant__ int steps_per_period = 2000;
__constant__ int periods = 1;
__constant__ int number_of_threads = 4;
__constant__ int afterstep_every = 1;

__constant__ float dt = 0.0020949113096826;

__shared__ float data[4][12];

__global__ void initkernel(int seed) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState_t* s = new hiprandState_t;
  hiprand_init(seed, idx, 0, s);
  curand_states[idx] = s;
}

__device__ __inline__ void calc_avg(float& current_avg, float new_value,
                                    int current_step) {
  current_avg +=
      (new_value - current_avg) / (current_step % steps_per_period + 1);
}

extern "C" __global__ void prepare_simulation(float* summary, float* output) {
  int idx = threadIdx.x + threadIdx.y * 2;

  data[idx][0] = 0.0f;  // current step

  float t = data[idx][1] = 0.0;  // t
  float v = data[idx][2] = 1.0;  // v
  data[idx][3] = 0.0f;           // avg_period_v
  data[idx][4] = 0.0f;           // avg_periods_v
  float x = data[idx][5] = 0.0;  // x
  data[idx][6] = 0.0f;           // avg_period_x
  data[idx][7] = 0.0f;           // avg_periods_x

  data[idx][8] =
      -0.100694794079146 * v + 0.201389588158292 * M_PI * sinf(2 * M_PI * x) +
      0.415366025576478 * cosf(3.749 * t) + 0.100694794079146;  // rhs

}

__device__ void afterstep(float v, float x, float rhs) {}

extern "C" __global__ void continue_simulation(float* summary, float* output) {
  int idx = threadIdx.x + threadIdx.y * 2;

  int current_step = (int) data[idx][0];
  float t = data[idx][1];
  float v = data[idx][2];
  float v_next;
  float avg_period_v = data[idx][3];
  float avg_periods_v = data[idx][4];
  float x = data[idx][5];
  float x_next;
  float avg_period_x = data[idx][6];
  float avg_periods_x = data[idx][7];
  float rhs = data[idx][8];
  float rhs_next;

  for (int i = 0; i < steps_per_kernel_call; i++) {
    /**
     * Averaging
     */
    // iterative mean https://stackoverflow.com/a/1934266/1185254
    calc_avg(avg_period_v, v, current_step);
    calc_avg(avg_period_x, x, current_step);

    if (current_step % (steps_per_period - 1) == 0) {
      calc_avg(avg_periods_v, avg_period_v, current_step / steps_per_period);
      avg_period_v = 0.0f;
      calc_avg(avg_periods_x, avg_period_x, current_step / steps_per_period);
      avg_period_x = 0.0f;
    }

    /**
	 * Integration
	 */
    // rand_val = curand_uniform(curand_states[idx]);

    v_next = v + rhs * dt;
    x_next = x + v * dt;
    rhs_next =
        -0.100694794079146 * v + 0.201389588158292 * M_PI * sinf(2 * M_PI * x) +
        0.415366025576478 * cosf(3.749 * t) + 0.100694794079146;

    t += dt;

    v = v_next;
    x = x_next;
    rhs = rhs_next;

    /**
	 * Afterstep
	 */
    if (current_step % 1 == 0) {
      afterstep(v_next, x_next, rhs_next);
    }

    current_step += 1;
  }

  data[idx][0] = current_step;
  data[idx][1] = t;
  data[idx][2] = v;
  data[idx][3] = avg_period_v;
  data[idx][4] = avg_periods_v;
  data[idx][5] = x;
  data[idx][6] = avg_period_x;
  data[idx][7] = avg_periods_x;
  data[idx][8] = rhs;
}

extern "C" __global__ void end_simulation(float* summary, float* output) {
  int idx = threadIdx.x + threadIdx.y * 2;

  summary[idx * 6 + 0] = data[idx][2];  // v
  summary[idx * 6 + 1] = data[idx][3];  // avg_period_v
  summary[idx * 6 + 2] = data[idx][4];  // avg_periods_v
  summary[idx * 6 + 3] = data[idx][5];  // x
  summary[idx * 6 + 4] = data[idx][6];  // avg_period_x
  summary[idx * 6 + 5] = data[idx][7];  // avg_periods_x
}